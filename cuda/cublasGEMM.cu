#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#include <sys/time.h>

const int M = 32;//A_ROW 
const int K = 16;//A_COL B_ROW
const int N = 32;//B_COL

double seconds(){
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char** argv){
    int dev = 0;
    hipDeviceProp_t deviceprop;
    hipGetDeviceProperties(&deviceprop, dev);
    hipSetDevice(dev);

    float* h_A = (float *)malloc(M*K*sizeof(float));
    float* h_B = (float *)malloc(K*N*sizeof(float));
    float* cpuref = (float *)malloc(M*N*sizeof(float));
    
    for(int i=0;i<M*K;i++){
        h_A[i] = i%10;
    }

    for(int i=0;i<M*K;i++){
        h_B[i] = i;
    }

    // for(int i=0;i<M*K;i++){
    //     printf("%f ", h_A[i]);
    //     if((i+1)%N==0) printf("\n");
    // }

    // printf("\n");

    // for(int i=0;i<K*N;i++){
    //     printf("%f ", h_B[i]);
    //     if((i+1)%N==0) printf("\n");
    // }

    printf("\n");
    float* d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, M*K*sizeof(float));
    hipMalloc((float **)&d_B, K*N*sizeof(float));
    hipMalloc((float **)&d_C, M*N*sizeof(float));

    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    double start = seconds();
    hipMemcpy(d_A, h_A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K*N*sizeof(float), hipMemcpyHostToDevice);

    float a = 1, b = 0;
    
    hipblasSgemm(
        blas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        N,
        M,
        K,
        &a,
        d_B,
        N,
        d_A,
        K,
        &b,
        d_C,
        N
    );

    hipMemcpy(cpuref, d_C, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    double Elpas = seconds() - start;
    printf("using %lf\n", Elpas);

    for(int i=0;i<16;++i){
        printf("%f ", cpuref[i]);
        if((i+1)%N==0) printf("\n");
    };
    printf("\n");

    free(cpuref);
    free(h_A);
    free(h_B);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();
    return 0;


}